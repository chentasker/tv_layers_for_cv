#include "hip/hip_runtime.h"
// Implements CUDA Kernel for DPTTRF

#include <torch/extension.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>


template <typename scalar_t>
__global__ void dual_to_primal_cont_p_cuda_kernel(
  torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> w,
  torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> x,
  torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> y,
  torch::PackedTensorAccessor<bool, 1,torch::RestrictPtrTraits,size_t> cont_update,
  long batch_size, long vec_size, long d_size)
{
  // Batch index
  const int b_idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Column index
  const int v_idx = blockIdx.y * blockDim.y + threadIdx.y;
  const int d_idx = blockIdx.z;

  // Check within range
  if(b_idx < batch_size && cont_update[b_idx] && v_idx <= vec_size && d_idx < d_size)
  {
    if (v_idx == 0)
      x[b_idx][0][d_idx] = y[b_idx][0]+w[b_idx][0][d_idx];
    else if (v_idx == vec_size)
      x[b_idx][vec_size][d_idx] = y[b_idx][vec_size]-w[b_idx][vec_size-1][d_idx];
    else
      x[b_idx][v_idx][d_idx] = y[b_idx][v_idx]-w[b_idx][v_idx-1][d_idx]+w[b_idx][v_idx][d_idx];
  }
}


void dual_to_primal_cont_p_cuda(torch::Tensor w, torch::Tensor x,
                                torch::Tensor y,
                                torch::Tensor cont_update
                               )
{
  const auto batch_size = x.size(0);
  const auto vec_size = x.size(1);
  const auto d_size = x.size(2);
  const auto nn = vec_size-1;

  const int threads = 32;
  const dim3 DimGrid(32, 32);
  const dim3 DimBlock((batch_size + threads - 1) / threads, (vec_size + threads - 1) / threads, d_size);

  AT_DISPATCH_FLOATING_TYPES(w.type(), "dual_to_primal_cont_p_cuda", ([&]
  {
    dual_to_primal_cont_p_cuda_kernel<scalar_t><<<DimBlock, DimGrid>>>(
      w.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
      x.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
      y.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
      cont_update.packed_accessor<bool,1,torch::RestrictPtrTraits,size_t>(),
      batch_size, nn, d_size);
  }));
}
