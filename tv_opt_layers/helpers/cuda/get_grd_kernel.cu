#include "hip/hip_runtime.h"
// Implements CUDA Kernel for DPTTRF

#include <torch/extension.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>


template <typename scalar_t>
__global__ void get_grd_cuda_kernel(
  torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> grd,
  torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> g,
  torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> d,
  torch::PackedTensorAccessor<long, 2,torch::RestrictPtrTraits,size_t> inactive,
  torch::PackedTensorAccessor<long, 1,torch::RestrictPtrTraits,size_t> num_inactive,
  long batch_size, long vec_size) {
    // Batch index
    const int b_idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Column index
    const int v_idx = blockIdx.y;
    
    // Check within range
    if(b_idx < batch_size && v_idx < vec_size) {
      const auto e_idx = num_inactive[b_idx];
      if (v_idx < e_idx)
        grd[b_idx][v_idx] = g[b_idx][inactive[b_idx][v_idx]]*d[b_idx][v_idx];
    }
}

void get_grd_cuda(torch::Tensor grd, torch::Tensor g,
                       torch::Tensor d,
                       torch::Tensor inactive,
                       torch::Tensor num_inactive) {
  const auto batch_size = grd.size(0);
  const auto vec_size = grd.size(1);

  const int threads = 1024;
  const dim3 blocks((batch_size + threads - 1) / threads, vec_size);

  AT_DISPATCH_FLOATING_TYPES(grd.type(), "get_grd_cuda", ([&] {
    get_grd_cuda_kernel<scalar_t><<<blocks, threads>>>(
      grd.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
      g.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(), 
      d.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
      inactive.packed_accessor<long,2,torch::RestrictPtrTraits,size_t>(), 
      num_inactive.packed_accessor<long,1,torch::RestrictPtrTraits,size_t>(), 
      batch_size, vec_size);
  }));
}