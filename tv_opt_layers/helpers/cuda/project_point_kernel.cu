#include "hip/hip_runtime.h"
// Implements CUDA Kernel for DPTTRF

#include <torch/extension.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>


template <typename scalar_t>
__global__ void project_point_cuda_kernel(
  torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> aux,
  torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> w,
  torch::PackedTensorAccessor<scalar_t,2,torch::RestrictPtrTraits,size_t> d,
  torch::PackedTensorAccessor<scalar_t,1,torch::RestrictPtrTraits,size_t> delta,
  torch::PackedTensorAccessor<long, 2,torch::RestrictPtrTraits,size_t> inactive,
  torch::PackedTensorAccessor<long, 1,torch::RestrictPtrTraits,size_t> num_inactive,
  torch::PackedTensorAccessor<bool, 1,torch::RestrictPtrTraits,size_t> cont_find,
  long batch_size, long vec_size)
{
  // Batch index
  const int b_idx = blockIdx.x * blockDim.x + threadIdx.x;
  // Column index
  const int v_idx = blockIdx.y;
  // Check within range
  if(b_idx < batch_size && v_idx < vec_size && cont_find[b_idx])
  {
    const auto e_idx = num_inactive[b_idx];
    if (v_idx < e_idx)
    {
      const auto ind = inactive[b_idx][v_idx];
      aux[b_idx][ind] = w[b_idx][ind] - delta[b_idx]*d[b_idx][v_idx];
    }
  }
}


void project_point_cuda(torch::Tensor aux, torch::Tensor w,
                        torch::Tensor d, torch::Tensor delta,
                        torch::Tensor inactive,
                        torch::Tensor num_inactive, torch::Tensor cont_find)
{
  const auto batch_size = aux.size(0);
  const auto vec_size = aux.size(1);

  const int threads = 1024;
  const dim3 blocks((batch_size + threads - 1) / threads, vec_size);

  AT_DISPATCH_FLOATING_TYPES(aux.type(), "project_point_cuda", ([&]
  {
    project_point_cuda_kernel<scalar_t><<<blocks, threads>>>(
      aux.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
      w.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
      d.packed_accessor<scalar_t,2,torch::RestrictPtrTraits,size_t>(),
      delta.packed_accessor<scalar_t,1,torch::RestrictPtrTraits,size_t>(),
      inactive.packed_accessor<long,2,torch::RestrictPtrTraits,size_t>(),
      num_inactive.packed_accessor<long,1,torch::RestrictPtrTraits,size_t>(),
      cont_find.packed_accessor<bool,1,torch::RestrictPtrTraits,size_t>(),
      batch_size, vec_size);
  }));
}
